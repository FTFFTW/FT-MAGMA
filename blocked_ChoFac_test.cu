#include "hip/hip_runtime.h"
/*Blocked Cholesky Factorization with Fault tolerance.
 *potf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
 *Jieyang Chen, University of California, Riverside
 **/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"printHelper.h"
#include"matrixGenerator.h"
#include"dpotrfFT.h"
#include"dtrsmFT.h"
#include"dsyrkFT.h"
#include"dgemmFT.h"
#include"checksumGenerator.h"


#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;


void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,
		float * real_time, float * proc_time, long long * flpins,
		float * mflops) {
	//cout<<"start my_dpotrf"<<endl;
	//initial data
	//int b_size = B;
	double * temp;
	//float gemm_time =0;
	//float cpu_time =0;
	hipHostAlloc((void**) &temp, B * B * sizeof(double), hipHostMallocDefault);
	//cout<<"pinned memory initialized"<<endl;
	//intial streams----------------------------
	hipStream_t stream0;  //for main loop
	hipStream_t stream1;  //for dgemm part
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//cout<<"Streams initialized"<<endl;
	//intial cublas
	hipblasStatus_t hipblasStatus_t;
	//hipblasHandle_t handle0;
	//hipblasStatus_t = hipblasCreate(&handle0);
	//if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	//  cout<<"CUBLAS NOT INITIALIZED(handle0)"<<endl;
	//hipblasStatus_t = hipblasSetStream(handle0,stream0);
	//if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	//  cout<<"CUBLAS SET STREAM NOT INITIALIZED(handle0)"<<endl;

	hipblasHandle_t handle1;
	hipblasStatus_t = hipblasCreate(&handle1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS NOT INITIALIZED(handle1) in my_dpotrf " << endl;

	hipblasStatus_t = hipblasSetStream(handle1, stream1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS SET STREAM NOT INITIALIZED(handle1) in my_dpotrf"
				<< endl;   

	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	
	//intialize checksum1 and checksum2
	double * v1=new double[B];
	double * v2=new double[B];
	for(int i=0;i<B;i++){
		v1[i]=1;
		v2[i]=i;
	}
	double * checksum1=initializeChecksum(handle1, matrix, ld, N, B, v1);
	double * checksum2=initializeChecksum(handle1, matrix, ld, N, B, v2);

	
	for (int i = 0; i < N; i += B) {
		//b_size = min(B,N-i);
		//cout<<"block size:"<<b_size<<"  ";
		

		if (i > 0) {

			//prepare for checkA

			double alpha = -1;
			double beta = 1;
			//hipEventRecord(start0,stream0);
			hipblasDsyrk(handle1, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, B, i,
					&alpha, matrix + i * ld, ld, &beta, matrix + i * ld + i,
					ld);
			//hipEventRecord(stop0,stream0);
		}
		
		hipStreamSynchronize(stream1);
		//hipEventRecord(start0,stream0);
		//hipHostAlloc((void**)&temp,b_size*b_size*sizeof(double),hipHostMallocDefault);
		hipMemcpy2DAsync(temp, B * sizeof(double), matrix + i * ld + i,
				ld * sizeof(double), B * sizeof(double), B,
				hipMemcpyDeviceToHost, stream0);

		if (i != 0 && i + B < N) {
			double alpha = -1;
			double beta = 1;
			//hipEventRecord(start1,stream1);                                                   
			hipblasDgemm(handle1, HIPBLAS_OP_T, HIPBLAS_OP_N, B, N - i - B, i,
					&alpha, matrix + i * ld, ld, matrix + (i + B) * ld, ld,
					&beta, matrix + (i + B) * ld + i, ld);
			//hipEventRecord(stop1,stream1);                                                    
		}
		hipStreamSynchronize(stream0);
		int info;
		dpotrf('U', B, temp, B, &info);
		hipMemcpy2DAsync(matrix + i * ld + i, ld * sizeof(double), temp,
				B * sizeof(double), B * sizeof(double), B,
				hipMemcpyHostToDevice, stream0);
		//hipEventRecord(stop0,stream0);

		/*if(i!=0&&i+b_size<ld){
		 hipEventSynchronize(stop1);
		 hipEventElapsedTime(&gemm_time,start1,stop1);
		 cout<<"GEMM: "<<gemm_time<<"ms  ";
		 }


		 hipEventSynchronize(stop0);
		 hipEventElapsedTime(&cpu_time,start0,stop0);
		 cout<<"CPU: "<<cpu_time<<"ms  "<<endl;
		 */
		//update B                                                                      
		if (i + B < N) {
			//hipStreamSynchronize(stream1);
			hipStreamSynchronize(stream0);
			double alpha = 1;
			//hipEventRecord(start0,stream0);
			hipblasDtrsm(handle1, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER,
					HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, B, N - i - B, &alpha,
					matrix + i * ld + i, ld, matrix + (i + B) * ld + i, ld);
			/*hipEventRecord(stop0,stream0);
			 hipEventSynchronize(stop0);
			 hipEventElapsedTime(&t,start0,stop0);
			 cout<<"TRSM: "<<t<<"ms  "<<endl;*/
		}
		//hipHostFree(temp);
	}
	//  t=clock()-t;
	//  float time =((float)t/CLOCKS_PER_SEC);
	//  cout<<"time[N="<<N<<"B="<<B<<"]:"<<time<<"s."<<endl;

	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	//  hipblasDestroy(handle0);
	hipblasDestroy(handle1);
	hipHostFree(temp);
	PAPI_shutdown();

}

void test_mydpotrf(int N, int B, float * real_time, float * proc_time,
		long long * flpins, float * mflops) {

	char uplo = 'u';
	double * matrix;
	double * result;
	size_t matrix_pitch;
	size_t result_pitch;
	//Memory allocation on RAM and DRAM
	hipMallocPitch((void**) &matrix, &matrix_pitch, N * sizeof(double), N);
	hipMallocPitch((void**) &result, &result_pitch, N * sizeof(double), N);

	int matrix_ld = matrix_pitch / sizeof(double);
	int result_ld = result_pitch / sizeof(double);

	matrixGenerator_gpu(uplo, matrix, matrix_ld, result, result_ld, N, 2);

	my_dpotrf(uplo, matrix, matrix_ld, N, B, real_time, proc_time, flpins,
			mflops);

	//Verify result
	//if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
	//cout<<"Result passed!"<<endl;
	//}else{
	//  cout<<"Result failed!"<<endl;
	// }

	hipFree(matrix);
	hipFree(result);

}

int main(int argc, char**argv) {

	int TEST_NUM = 1;
	int n[10] = { 16, 2048, 3072, 4096, 5120, 6144, 7168, 8192, 9216, 10240 };
	int b = 2;
	for (int k = 0; k < 1; k++) {
		float total_real_time = 0.0;
		float total_proc_time = 0.0;
		long long total_flpins = 0.0;
		float total_mflops = 0.0;
		float real_time = 0.0;
		float proc_time = 0.0;
		long long flpins = 0.0;
		float mflops = 0.0;
		double flops = FLOPS_DPOTRF(n[k]) / 1e9;
		//cout<<"flops:"<<flops<<"  ";

		for (int i = 0; i < TEST_NUM; i++) {
			test_mydpotrf(n[k], b, &real_time, &proc_time, &flpins, &mflops);
			total_real_time += real_time;
			total_proc_time += proc_time;
			total_flpins += flpins;
			total_mflops += mflops;
		}
		cout << "Size:" << n[k] << "(" << b << ")---Real_time:"
				<< total_real_time / (double) TEST_NUM << "---" << "Proc_time:"
				<< total_proc_time / (double) TEST_NUM << "---"
				<< "Total GFlops:"
				<< flops / (total_proc_time / (double) TEST_NUM) << endl;
	}
}
