/*Blocked Cholesky Factorization with Fault tolerance.
dpotf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
*Jieyang Chen, University of California, Riverside
**/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"printHelper.h"
#include"matrixGenerator.h"
#include"dpotrfFT.h"
#include"dtrsmFT.h"
#include"dsyrkFT.h"
#include"dgemmFT.h"
#include"checksumGenerator.h"
#include"hip/hip_runtime_api.h"

#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;

void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,
		float * real_time, float * proc_time, long long * flpins,
		float * mflops, bool FT) {

	double * temp;
	hipHostAlloc((void**) &temp, B * B * sizeof(double), hipHostMallocDefault);

	//intial streams----------------------------
	hipStream_t stream0;  //for main loop
	hipStream_t stream1;  //for dgemm part
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//intial cublas
	hipblasStatus_t hipblasStatus_t;
	hipblasHandle_t handle1;
	hipblasStatus_t = hipblasCreate(&handle1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS NOT INITIALIZED(handle1) in my_dpotrf " << endl;
	hipblasStatus_t = hipblasSetStream(handle1, stream1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS SET STREAM NOT INITIALIZED(handle1) in my_dpotrf"
				<< endl;

	//variables for FT
	double * v;
	int v_ld;
	//double * v2;
	double * vd;
	//double * v2d;
	size_t vd_pitch;
	int vd_ld;
	//size_t v2d_pitch;
	double * chk;
	//double * chk2;
	double * chkd;
	//double * chk2d;
	size_t chkd_pitch;
	//size_t chk2d_pitch;
	int chkd_ld;
	//int chk2d_ld;
	size_t checksum_pitch;
	//size_t checksum2_pitch;
	double * checksum;
	//double * checksum2;
	int checksum_ld;
	//int checksum2_ld;

	if (FT) {
		//cout<<"check sum initialization started"<<endl;
		//intialize checksum vector on CPU
		v = new double[B * 2];
		//v2 = new double[B];
		//first vector
		for (int i = 0; i < B; ++i) {
			*(v + i) = 1;
		}
		for (int i = 0; i < B; ++i) {
			*(v + i + B) = i+1;
		}
		v_ld = B;
		printMatrix_host(v, B, 2);
		
		//cout<<"checksum vector on CPU initialized"<<endl;

		//intialize checksum vector on GPU
		hipMallocPitch((void**) &vd, &vd_pitch, B * sizeof(double), 2);
		vd_ld = vd_pitch / sizeof(double);
		hipMemcpy2D(vd, vd_pitch, v, B * sizeof(double), B * sizeof(double),
				2, hipMemcpyHostToDevice);
		
		printMatrix_gpu(vd, vd_pitch, B, 2);
		//cout<<"checksum vector on gpu initialized"<<endl;

		
		
		//allocate space for recalculated checksum on CPU
		chk = new double[B * 2];
		//cout<<"allocated space for recalculated checksum on CPU"<<endl;

		//allocate space for reclaculated checksum on CPU
		hipMallocPitch((void**) &chkd, &chkd_pitch, (N / B) * 2 * sizeof(double),B);
		chkd_ld = chkd_pitch / sizeof(double);
		//cout<<"allocated space for recalculated checksum on GPU"<<endl;

		//initialize checksums
		checksum = initializeChecksum(handle1, matrix, ld, N, B, vd, vd_ld, checksum_pitch);
		checksum_ld = checksum_pitch / sizeof(double);
		printMatrix_gpu(checksum, checksum_pitch, (N/B)*2, N);
		//cout<<"checksums initialized"<<endl;

	}
	
	
	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	//start of profiling
	hipProfilerStart();
	
	for (int i = 0; i < N; i += B) {

		 //cout<<"i="<<i<<endl;
		 //cout<<"matrix:"<<endl;
		 //printMatrix_gpu(matrix, ld*sizeof(double), N, N);
		 //cout<<"checksum:"<<endl;
		 //printMatrix_gpu(checksum, checksum_pitch, (N/B)*2, N);
		 
		if (i > 0) {
			dsyrkFT(handle1, B, i, matrix + i, ld, matrix + i * ld + i, ld,
					checksum1 + i / B, checksum1_ld, checksum2 + i / B,
					checksum2_ld, checksum1 + (i / B) + i * checksum1_ld,
					checksum1_ld, checksum2 + (i / B) + i * checksum2_ld,
					checksum2_ld, v1d, v2d, chk1d, chk1d_ld, chk2d, chk2d_ld,
					FT);
			
		}
		
		
		hipStreamSynchronize(stream1);
		
		hipMemcpy2DAsync(temp, B * sizeof(double), matrix + i * ld + i,
				ld * sizeof(double), B * sizeof(double), B,
				hipMemcpyDeviceToHost, stream0);
		
		
		if (FT) {
			 hipMemcpy2DAsync(chk, 2 * sizeof(double), checksum + (i/B) * 2 + i*checksum_ld,
			 checksum_pitch, 2 * sizeof(double), B,
			 hipMemcpyDeviceToHost, stream0);
			 //hipMemcpy2DAsync(chk2, 1 * sizeof(double), checksum2 + (i/B) + i*checksum2_ld,
			 //checksum2_pitch, 1 * sizeof(double), B,
			 //hipMemcpyDeviceToHost, stream0);
			 
		}
		/*
		if (i != 0 && i + B < N) {

			dgemmFT(handle1, N - i - B, B, i, matrix + (i + B), ld, matrix + i,
					ld, matrix + i * ld + (i + B), ld, checksum1 + (i + B) / B,
					checksum1_ld, checksum2 + (i + B) / B, checksum2_ld,
					checksum1 + i * checksum1_ld + (i + B) / B, checksum1_ld,
					checksum2 + i * checksum2_ld + (i + B) / B, checksum2_ld,
					v1d, v2d, chk1d, chk1d_ld, chk2d, chk2d_ld, FT);
		}
		
		
		hipStreamSynchronize(stream0);
		*/
		//int info;
		dpotrfFT(temp, B, B, chk, 2, v, v_ld, FT);
		
		hipMemcpy2DAsync(matrix + i * ld + i, ld * sizeof(double), temp,
				B * sizeof(double), B * sizeof(double), B,
				hipMemcpyHostToDevice, stream0);
		
		
		if (FT) {
			 hipMemcpy2DAsync(checksum + (i/B) * 2 + i*checksum_ld, checksum_pitch, chk, 2 * sizeof(double), 
			 2 * sizeof(double), B,
			 hipMemcpyHostToDevice, stream0);
			 //hipMemcpy2DAsync(checksum2 + (i/B) + i*checksum2_ld,checksum2_pitch, chk2, 1 * sizeof(double), 
			 //1 * sizeof(double), B,
			 //hipMemcpyHostToDevice, stream0);
			 
		}
		
		
		//update B    
		
		if (i + B < N) {
			hipStreamSynchronize(stream0);
			dtrsmFT(handle1, N - i - B, B, matrix + i * ld + i, ld,
					matrix + i * ld + i + B, ld,
					checksum + ((i + B) / B )*2 + i * checksum_ld, checksum_ld,
					vd, vd_ld, chkd, chkd_ld, FT);
		}
		
		

	}

	
	//end of profiling
	hipProfilerStop();

	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipblasDestroy(handle1);
	hipHostFree(temp);
	PAPI_shutdown();

}

void test_mydpotrf(int N, int B, float * real_time, float * proc_time,
		long long * flpins, float * mflops, bool FT) {

	char uplo = 'l';
	double * matrix;
	double * result;
	size_t matrix_pitch;
	size_t result_pitch;
	//Memory allocation on RAM and DRAM
	hipMallocPitch((void**) &matrix, &matrix_pitch, N * sizeof(double), N);
	hipMallocPitch((void**) &result, &result_pitch, N * sizeof(double), N);

	int matrix_ld = matrix_pitch / sizeof(double);
	int result_ld = result_pitch / sizeof(double);

	matrixGenerator_gpu2(uplo, matrix, matrix_ld, result, result_ld, N, 2);
	//hipFree(result);

	my_dpotrf(uplo, matrix, matrix_ld, N, B, real_time, proc_time, flpins,
			mflops, FT);

	//Verify result
	/*if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
	 cout<<"Result passed!"<<endl;
	 }else{
	 cout<<"Result failed!"<<endl;
	 }
	 */
	hipFree(matrix);
	//hipFree(result);

}

int main(int argc, char**argv) {
	int N = atoi(argv[1]);
	int B = atoi(argv[2]);
	bool FT = false;
	if (argv[3][0] == '1')
		FT = true;
	int TEST_NUM = 1;
	cout << "Input config:N=" << N << ", B=" << B << ", FT=" << FT << endl;
	//int n[10] = { 1024, 2048, 3072, 4096, 5120, 6144, 7168, 8192, 9216, 10240 };
	//int b = 256; 
	//for (int k = 0; k < 1; k++) {
	float total_real_time = 0.0;
	float total_proc_time = 0.0;
	long long total_flpins = 0.0;
	float total_mflops = 0.0;
	float real_time = 0.0;
	float proc_time = 0.0;
	long long flpins = 0.0;
	float mflops = 0.0;
	double flops = FLOPS_DPOTRF(N) / 1e9;
	//cout<<"flops:"<<flops<<"  ";

	for (int i = 0; i < TEST_NUM; i++) {
		test_mydpotrf(N, B, &real_time, &proc_time, &flpins, &mflops, FT);
		total_real_time += real_time;
		total_proc_time += proc_time;
		total_flpins += flpins;
		total_mflops += mflops;
	}
	if (FT)
		cout << "FT enabled" << endl;
	cout << "Size:" << N << "(" << B << ")---Real_time:"
			<< total_real_time / (double) TEST_NUM << "---" << "Proc_time:"
			<< total_proc_time / (double) TEST_NUM << "---" << "Total GFlops:"
			<< flops / (total_proc_time / (double) TEST_NUM) << endl;
	hipDeviceReset();
}
