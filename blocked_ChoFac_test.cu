/*Blocked Cholesky Factorization with Fault tolerance.
dpotf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
*Jieyang Chen, University of California, Riverside
**/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"printHelper.h"
#include"matrixGenerator.h"
#include"dpotrfFT.h"
#include"dtrsmFT.h"
#include"dsyrkFT.h"
#include"dgemmFT.h"
#include"checksumGenerator.h"
#include"hip/hip_runtime_api.h"

#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;

void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,
		float * real_time, float * proc_time, long long * flpins,
		float * mflops, bool FT) {

	double * temp;
	hipHostAlloc((void**) &temp, B * B * sizeof(double), hipHostMallocDefault);

	//intial streams----------------------------
	hipStream_t stream0;  //for main loop
	hipStream_t stream1;  //for dgemm part
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//intial cublas
	hipblasStatus_t hipblasStatus_t;
	hipblasHandle_t handle1;
	hipblasStatus_t = hipblasCreate(&handle1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS NOT INITIALIZED(handle1) in my_dpotrf " << endl;
	hipblasStatus_t = hipblasSetStream(handle1, stream1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS SET STREAM NOT INITIALIZED(handle1) in my_dpotrf"
				<< endl;

	//variables for FT
	double * v1;
	double * v2;
	double * v1d;
	double * v2d;
	size_t v1d_pitch;
	size_t v2d_pitch;
	double * chk1;
	double * chk2;
	double * chk1d;
	double * chk2d;
	size_t chk1d_pitch;
	size_t chk2d_pitch;
	int chk1d_ld;
	int chk2d_ld;
	size_t checksum1_pitch;
	size_t checksum2_pitch;
	double * checksum1;
	double * checksum2;
	int checksum1_ld;
	int checksum2_ld;

	if (FT) {
		//cout<<"check sum initialization started"<<endl;
		//intialize checksum vector on CPU
		v1 = new double[B];
		v2 = new double[B];
		for (int i = 0; i < B; i++) {
			v1[i] = 1;
			v2[i] = i + 1;
		}
		//cout<<"checksum vector on CPU initialized"<<endl;

		//intialize checksum vector on GPU
		hipMallocPitch((void**) &v1d, &v1d_pitch, B * sizeof(double), 1);
		hipMemcpy2D(v1d, v1d_pitch, v1, B * sizeof(double), B * sizeof(double),
				1, hipMemcpyHostToDevice);
		hipMallocPitch((void**) &v2d, &v2d_pitch, B * sizeof(double), 1);
		hipMemcpy2D(v2d, v2d_pitch, v2, B * sizeof(double), B * sizeof(double),
				1, hipMemcpyHostToDevice);
		//cout<<"checksum vector on gpu initialized"<<endl;

		//allocate space for recalculated checksum on CPU
		chk1 = new double[B];
		chk2 = new double[B];
		//cout<<"allocate space for recalculated checksum on CPU"<<endl;

		//allocate space for reclaculated checksum on CPU
		hipMallocPitch((void**) &chk1d, &chk1d_pitch, (N / B) * sizeof(double),
				B);
		hipMallocPitch((void**) &chk2d, &chk2d_pitch, (N / B) * sizeof(double),
				B);
		chk1d_ld = chk1d_pitch / sizeof(double);
		chk2d_ld = chk2d_pitch / sizeof(double);
		//cout<<"allocate space for recalculated checksum on GPU"<<endl;

		//initialize checksums
		checksum1 = initializeChecksum(handle1, matrix, ld, N, B, v1d,
				checksum1_pitch);
		checksum2 = initializeChecksum(handle1, matrix, ld, N, B, v2d,
				checksum2_pitch);
		checksum1_ld = checksum1_pitch / sizeof(double);
		checksum2_ld = checksum2_pitch / sizeof(double);
		//cout<<"checksums initialized"<<endl;

	}

	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	//start of profiling
	hipProfilerStart();

	for (int i = 0; i < N; i += B) {

		/*cout<<"i="<<i<<endl;
		 cout<<"matrix:"<<endl;
		 printMatrix_gpu(matrix, ld*sizeof(double), N, N);
		 cout<<"checksum1:"<<endl;
		 printMatrix_gpu(checksum1, checksum1_pitch, N/B, N);
		 cout<<"checksum2:"<<endl;
		 printMatrix_gpu(checksum2, checksum2_pitch, N/B, N);
		 */
		if (i > 0) {
			dsyrkFT(handle1, B, i, matrix + i, ld, matrix + i * ld + i, ld,
					checksum1 + i / B, checksum1_ld, checksum2 + i / B,
					checksum2_ld, checksum1 + (i / B) + i * checksum1_ld,
					checksum1_ld, checksum2 + (i / B) + i * checksum2_ld,
					checksum2_ld, v1d, v2d, chk1d, chk1d_ld, chk2d, chk2d_ld,
					FT);

			/*hipblasDsyrk(handle1, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, B, i,
			 &negone, matrix + i, ld, &one, matrix + i * ld + i,
			 ld);
			 */
		}

		hipStreamSynchronize(stream1);

		hipMemcpy2DAsync(temp, B * sizeof(double), matrix + i * ld + i,
				ld * sizeof(double), B * sizeof(double), B,
				hipMemcpyDeviceToHost, stream0);

		if (FT) {
			 hipMemcpy2DAsync(chk1, 1 * sizeof(double), checksum1 + (i/B) + i*checksum1_ld,
			 checksum1_pitch, 1 * sizeof(double), B,
			 hipMemcpyDeviceToHost, stream0);
			 hipMemcpy2DAsync(chk2, 1 * sizeof(double), checksum2 + (i/B) + i*checksum2_ld,
			 checksum2_pitch, 1 * sizeof(double), B,
			 hipMemcpyDeviceToHost, stream0);
			 
		}

		if (i != 0 && i + B < N) {

			dgemmFT(handle1, N - i - B, B, i, matrix + (i + B), ld, matrix + i,
					ld, matrix + i * ld + (i + B), ld, checksum1 + (i + B) / B,
					checksum1_ld, checksum2 + (i + B) / B, checksum2_ld,
					checksum1 + i * checksum1_ld + (i + B) / B, checksum1_ld,
					checksum2 + i * checksum2_ld + (i + B) / B, checksum2_ld,
					v1d, v2d, chk1d, chk1d_ld, chk2d, chk2d_ld, FT);

			/*hipblasDgemm(handle1, HIPBLAS_OP_N, HIPBLAS_OP_T, N - i - B, B, i,
			 &negone, matrix + (i + B), ld, matrix + i, ld,
			 &one, matrix + i * ld + (i + B), ld);
			 */
		}
		hipStreamSynchronize(stream0);

		//int info;
		//dpotrf('L', B, temp, B, &info);
		dpotrfFT(temp, B, B, chk1, 1, chk2, 1, v1, v2, FT);

		hipMemcpy2DAsync(matrix + i * ld + i, ld * sizeof(double), temp,
				B * sizeof(double), B * sizeof(double), B,
				hipMemcpyHostToDevice, stream0);
		if (FT) {
			hipMemcpy2DAsync(checksum1 + (i/B) + i*checksum1_ld,checksum1_pitch, chk1, 1 * sizeof(double), 
			 1 * sizeof(double), B,
			 hipMemcpyHostToDevice, stream0);
			 hipMemcpy2DAsync(checksum2 + (i/B) + i*checksum2_ld,checksum2_pitch, chk2, 1 * sizeof(double), 
			 1 * sizeof(double), B,
			 hipMemcpyHostToDevice, stream0);
			 
		}

		//update B                                                                      
		if (i + B < N) {
			hipStreamSynchronize(stream0);
			dtrsmFT(handle1, N - i - B, B, matrix + i * ld + i, ld,
					matrix + i * ld + i + B, ld,
					checksum1 + (i + B) / B + i * checksum1_ld, checksum1_ld,
					checksum2 + (i + B) / B + i * checksum2_ld, checksum2_ld,
					v1d, v2d, chk1d, chk1d_ld, chk2d, chk2d_ld, FT);
			/*hipblasDtrsm(handle1, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
			 HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, N - i - B, B,  &one,
			 matrix + i * ld + i, ld, matrix + i * ld + i + B, ld);
			 */
		}

	}

	//end of profiling
	hipProfilerStop();

	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipblasDestroy(handle1);
	hipHostFree(temp);
	PAPI_shutdown();

}

void test_mydpotrf(int N, int B, float * real_time, float * proc_time,
		long long * flpins, float * mflops, bool FT) {

	char uplo = 'l';
	double * matrix;
	//double * result;
	size_t matrix_pitch;
	//size_t result_pitch;
	//Memory allocation on RAM and DRAM
	hipMallocPitch((void**) &matrix, &matrix_pitch, N * sizeof(double), N);
	//hipMallocPitch((void**) &result, &result_pitch, N * sizeof(double), N);

	int matrix_ld = matrix_pitch / sizeof(double);
	//int result_ld = result_pitch / sizeof(double);

	matrixGenerator_gpu(uplo, matrix, matrix_ld, N, 2);
	//hipFree(result);

	my_dpotrf(uplo, matrix, matrix_ld, N, B, real_time, proc_time, flpins,
			mflops, FT);

	//Verify result
	/*if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
	 cout<<"Result passed!"<<endl;
	 }else{
	 cout<<"Result failed!"<<endl;
	 }
	 */
	hipFree(matrix);
	//hipFree(result);

}

int main(int argc, char**argv) {
	int N = atoi(argv[1]);
	int B = atoi(argv[2]);
	bool FT = false;
	if (argv[3][0] == '1')
		FT = true;
	int TEST_NUM = 5;
	cout << "Input config:N=" << N << ", B=" << B << ", FT=" << FT << endl;
	//int n[10] = { 1024, 2048, 3072, 4096, 5120, 6144, 7168, 8192, 9216, 10240 };
	//int b = 256; 
	//for (int k = 0; k < 1; k++) {
	float total_real_time = 0.0;
	float total_proc_time = 0.0;
	long long total_flpins = 0.0;
	float total_mflops = 0.0;
	float real_time = 0.0;
	float proc_time = 0.0;
	long long flpins = 0.0;
	float mflops = 0.0;
	double flops = FLOPS_DPOTRF(N) / 1e9;
	//cout<<"flops:"<<flops<<"  ";

	for (int i = 0; i < TEST_NUM; i++) {
		test_mydpotrf(N, B, &real_time, &proc_time, &flpins, &mflops, FT);
		total_real_time += real_time;
		total_proc_time += proc_time;
		total_flpins += flpins;
		total_mflops += mflops;
	}
	if (FT)
		cout << "FT enabled" << endl;
	cout << "Size:" << N << "(" << B << ")---Real_time:"
			<< total_real_time / (double) TEST_NUM << "---" << "Proc_time:"
			<< total_proc_time / (double) TEST_NUM << "---" << "Total GFlops:"
			<< flops / (total_proc_time / (double) TEST_NUM) << endl;
	hipDeviceReset();
}
