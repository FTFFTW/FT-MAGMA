/*Blocked Cholesky Factorization with Fault tolerance.
 *potf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
 *Jieyang Chen, University of California, Riverside
 **/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"printHelper.h"
#include"matrixGenerator.h"
#include"dpotrfFT.h"
#include"dtrsmFT.h"
#include"dsyrkFT.h"
#include"dgemmFT.h"
#include"checksumGenerator.h"


#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;


void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,
		float * real_time, float * proc_time, long long * flpins,
		float * mflops) {
	
	
	double * temp;
	hipHostAlloc((void**) &temp, B * B * sizeof(double), hipHostMallocDefault);
	
	//intial streams----------------------------
	hipStream_t stream0;  //for main loop
	hipStream_t stream1;  //for dgemm part
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//cout<<"Streams initialized"<<endl;
	//intial cublas
	hipblasStatus_t hipblasStatus_t;
	hipblasHandle_t handle1;
	hipblasStatus_t = hipblasCreate(&handle1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS NOT INITIALIZED(handle1) in my_dpotrf " << endl;
	hipblasStatus_t = hipblasSetStream(handle1, stream1);
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		cout << "CUBLAS SET STREAM NOT INITIALIZED(handle1) in my_dpotrf"
				<< endl;   

	
	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	
	//intialize checksum1 and checksum2
	double * v1=new double[B];
	double * v2=new double[B];
	for(int i=0;i<B;i++){
		v1[i]=1;
		v2[i]=i;
	}
	size_t checksum1_pitch;
	size_t checksum2_pitch;
	double * checksum1=initializeChecksum(handle1, matrix, ld, N, B, v1, checksum1_pitch);
	double * checksum2=initializeChecksum(handle1, matrix, ld, N, B, v2, checksum2_pitch);
	cout<<"checksum1:"<<endl;
	printMatrix_gpu(checksum1, checksum1_pitch, N/B, N);
	cout<<"checksum2:"<<endl;
	printMatrix_gpu(checksum2, checksum2_pitch, N/B, N);

	double one = 1;
	double negone = -1;
	double zero = 0;
	
	for (int i = 0; i < N; i += B) {
		
		if (i > 0) {
			
			hipblasDsyrk(handle1, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, B, i,
					&negone, matrix + i, ld, &one, matrix + i * ld + i,
					ld);
			
		}
		
		hipStreamSynchronize(stream1);
		
		hipMemcpy2DAsync(temp, B * sizeof(double), matrix + i * ld + i,
				ld * sizeof(double), B * sizeof(double), B,
				hipMemcpyDeviceToHost, stream0);

		if (i != 0 && i + B < N) {
			                   
			hipblasDgemm(handle1, HIPBLAS_OP_N, HIPBLAS_OP_T, N - i - B, B, i,
					&negone, matrix + (i + B), ld, matrix + i, ld,
					&one, matrix + i * ld + (i + B), ld);
			                                                
		}
		hipStreamSynchronize(stream0);
		int info;
		dpotrf('L', B, temp, B, &info);
		hipMemcpy2DAsync(matrix + i * ld + i, ld * sizeof(double), temp,
				B * sizeof(double), B * sizeof(double), B,
				hipMemcpyHostToDevice, stream0);
	
		//update B                                                                      
		if (i + B < N) {
			hipStreamSynchronize(stream0);
		
			hipblasDtrsm(handle1, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
					HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, N - i - B, B,  &one,
					matrix + i * ld + i, ld, matrix + i * ld + i + B, ld);
	
		}

	}
	
	if (PAPI_flops(real_time, proc_time, flpins, mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	
	hipblasDestroy(handle1);
	hipHostFree(temp);
	PAPI_shutdown();

}

void test_mydpotrf(int N, int B, float * real_time, float * proc_time,
		long long * flpins, float * mflops) {

	char uplo = 'l';
	double * matrix;
	double * result;
	size_t matrix_pitch;
	size_t result_pitch;
	//Memory allocation on RAM and DRAM
	hipMallocPitch((void**) &matrix, &matrix_pitch, N * sizeof(double), N);
	hipMallocPitch((void**) &result, &result_pitch, N * sizeof(double), N);

	int matrix_ld = matrix_pitch / sizeof(double);
	int result_ld = result_pitch / sizeof(double);

	matrixGenerator_gpu(uplo, matrix, matrix_ld, result, result_ld, N, 2);

	my_dpotrf(uplo, matrix, matrix_ld, N, B, real_time, proc_time, flpins,
			mflops);

	//Verify result
	if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
	cout<<"Result passed!"<<endl;
	}else{
	  cout<<"Result failed!"<<endl;
	}

	hipFree(matrix);
	hipFree(result);

}

int main(int argc, char**argv) {

	int TEST_NUM = 1;
	int n[10] = { 16, 2048, 3072, 4096, 5120, 6144, 7168, 8192, 9216, 10240 };
	int b = 2;
	for (int k = 0; k < 1; k++) {
		float total_real_time = 0.0;
		float total_proc_time = 0.0;
		long long total_flpins = 0.0;
		float total_mflops = 0.0;
		float real_time = 0.0;
		float proc_time = 0.0;
		long long flpins = 0.0;
		float mflops = 0.0;
		double flops = FLOPS_DPOTRF(n[k]) / 1e9;
		//cout<<"flops:"<<flops<<"  ";

		for (int i = 0; i < TEST_NUM; i++) {
			test_mydpotrf(n[k], b, &real_time, &proc_time, &flpins, &mflops);
			total_real_time += real_time;
			total_proc_time += proc_time;
			total_flpins += flpins;
			total_mflops += mflops;
		}
		cout << "Size:" << n[k] << "(" << b << ")---Real_time:"
				<< total_real_time / (double) TEST_NUM << "---" << "Proc_time:"
				<< total_proc_time / (double) TEST_NUM << "---"
				<< "Total GFlops:"
				<< flops / (total_proc_time / (double) TEST_NUM) << endl;
	}
}
