#include "hip/hip_runtime.h"

#include<stdio.h>
#include<iostream>
#include"papi.h"
#define NB 512

using namespace std;

// encoding checksum for A

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{
	//if (blockIdx.x == 0 && threadIdx.x == 0)
	//	printf("grid:%d, block:%d\n", gridDim.x, blockDim.x);
    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	int i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}



__global__ void
chkenc_kernel2(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * NB + threadIdx.x;

	A = A + idx * lda;

	double temp = 0;
	double temp2 = 0;
	for (int i = 0; i < NB; i++) {
		temp += A[i];
		temp2 += A[i] * (i+1);
	}
	*(Chk + idx * ldchk) = temp;
	*(Chk + idx * ldchk+1) = temp2;
	
}


void chkenc(double * A, int lda, int m, int n, double * Chk , int ldchk, hipStream_t stream) {
    int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 512; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel2, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	chkenc_kernel2<<<n, m/n, 0, stream>>>(A, lda, Chk, ldchk);
}

int main(){
	int n = 30720;
	double * A = new double[NB * n];
	for (int i = 0; i < NB*n; i++) {
		A[i] = i;
	}
	double * dA;
	size_t dApitch;
	hipMallocPitch(&dA, &dApitch, NB*sizeof(double), n);
	hipMemcpy2D(dA, dApitch, A, NB, NB, n, hipMemcpyHostToDevice);
	int ldda = dApitch/sizeof(double);

	double * chk;
	size_t chkpitch;
	hipMallocPitch(&chk, &chkpitch, 2*sizeof(double), n);
	int ldchk = chkpitch/sizeof(double);

	hipStream_t stream;
	hipStreamCreate(&stream);

	float real_time = 0.0;
	float proc_time = 0.0;
	long long flpins = 0.0;
	float mflops = 0.0;

	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	chkenc(dA, ldda, NB, n, chk , ldchk, stream);
	hipStreamSynchronize(stream);
	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}

	cout << real_time;






}



