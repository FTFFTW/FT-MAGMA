#include "hip/hip_runtime.h"

#include<stdio.h>
#include<iostream>
#include"papi.h"
#define N 30720
#define NB 512
#define rB 32
#define cB 32

using namespace std;

// encoding checksum for A

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	/* logrithm reduction */
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


__global__ void
chkenc_kernel1_5(double * A, int lda, double * Chk , int ldchk)
{

	//blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();


	double sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk) = sum;
	}

	__syncthreads();

	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk + 1) = sum;
	}

	
}





__global__ void
chkenc_kernel2(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * NB + threadIdx.x;

	A = A + idx * lda;

	double temp = 0;
	double temp2 = 0;
	for (int i = 0; i < NB; i++) {
		temp += A[i];
		temp2 += A[i] * (i+1);
	}
	*(Chk + idx * ldchk) = temp;
	*(Chk + idx * ldchk+1) = temp2;
	
}


__global__ void
chkenc_kernel3(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * cB;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	__shared__ double cache[rB][cB];

	for (int i = 0; i < NB; i += rB) {
		
		//load a block to cache
		for (int j = 0; j < rB; j++) {
			cache[threadIdx.x][j] = *(A + j * lda + threadIdx.x);
		}
		__syncthreads();

		for (int j = 0; j < rB; j++) {
			sum1 += cache[j][threadIdx.x];
			sum2 += cache[j][threadIdx.x] * (i + j + 1);
		}
		__syncthreads();
		A = A + rB;
	}

	*(Chk + idx * ldchk) = sum1;
	*(Chk + idx * ldchk+1) = sum2;
	
}

__global__ void
chkenc_kernel3_5(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * cB;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	__shared__ double cache[rB][cB];

	for (int i = 0; i < NB; i += rB) {
		
		//load a block to cache
		cache[threadIdx.x][threadIdx.y] = *(A + threadIdx.y * lda + threadIdx.x);
		__syncthreads();
		int k = rB / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.x][threadIdx.y] += cache[threadIdx.x + k][threadIdx.y];
			}
			
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum1 += cache[0][threadIdx.y];
		}

		cache[threadIdx.x][threadIdx.y] = *(A + threadIdx.y * lda + threadIdx.x) * (i + threadIdx.x + 1);
		__syncthreads();
		k = rB / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.x][threadIdx.y] += cache[threadIdx.x + k][threadIdx.y];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum2 += cache[0][threadIdx.y];
		}
				
		A = A + rB;
	}


	if (threadIdx.x == 0) {
		*(Chk + idx * ldchk) = sum1;
		*(Chk + idx * ldchk+1) = sum2;
	}
	
}


__global__ void
chkenc_kernel4(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    for(int k = 0; k < N; k += cB) {

	    double sum1 = 0;
	    double sum2 = 0;

		A = A + k * lda;

		__shared__ double cache[rB][cB];

		for (int i = 0; i < NB; i += rB) {
			
			//load a block to cache
			for (int j = 0; j < rB; j++) {
				cache[threadIdx.x][j] = *(A + j * lda + threadIdx.x);
			}
			__syncthreads();

			for (int j = 0; j < rB; j++) {
				sum1 += cache[j][threadIdx.x];
				sum2 += cache[j][threadIdx.x] * (i + j + 1);
			}
			__syncthreads();
			A = A + rB;
		}

		*(Chk + k * ldchk) = sum1;
		*(Chk + k * ldchk+1) = sum2;
	}
	
}


void chkenc(double * A, int lda, int m, int n, double * Chk , int ldchk, hipStream_t stream) {
    int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = cB; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel4, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	chkenc_kernel<<<N, NB, 0, stream>>>(A, lda, Chk, ldchk);

	//dim3 d(cB, rB, 1);
	//chkenc_kernel3_5<<<N/cB, d, 0, stream>>>(A, lda, Chk, ldchk);
	//chkenc_kernel4<<<1, cB, 0, stream>>>(A, lda, Chk, ldchk);
}

int main(){
	
	double * A = new double[NB * N];
	for (int i = 0; i < NB*N; i++) {
		A[i] = i;
	}
	double * dA;
	size_t dApitch;
	hipMallocPitch(&dA, &dApitch, NB*sizeof(double), N);
	hipMemcpy2D(dA, dApitch, A, NB, NB, N, hipMemcpyHostToDevice);
	int ldda = dApitch/sizeof(double);

	double * chk;
	size_t chkpitch;
	hipMallocPitch(&chk, &chkpitch, 2*sizeof(double), N);
	int ldchk = chkpitch/sizeof(double);

	hipStream_t stream;
	hipStreamCreate(&stream);

	float real_time = 0.0;
	float proc_time = 0.0;
	long long flpins = 0.0;
	float mflops = 0.0;
	long long flops = 2 * NB * N * 2;


	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}

	chkenc_kernel<<<N, NB, 0, stream>>>(dA, ldda, chk, ldchk);
	hipStreamSynchronize(stream);
	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	cout << real_time << "\t" << (flops/real_time)/1e9 << "\t";

	PAPI_shutdown();


	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	//chkenc(dA, ldda, NB, N, chk , ldchk, stream);
	
	chkenc_kernel<<<N, NB, 0, stream>>>(dA, ldda, chk, ldchk);
	hipStreamSynchronize(stream);
	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	
	cout << real_time << "\t" << (flops/real_time)/1e9 << "\t";



	real_time = 0.0;
	proc_time = 0.0;
	flpins = 0.0;
	mflops = 0.0;

	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	
	chkenc_kernel1_5<<<N, NB, 0, stream>>>(dA, ldda, chk, ldchk);
	hipStreamSynchronize(stream);
	if (PAPI_flops(&real_time, &proc_time, &flpins, &mflops) < PAPI_OK) {
		cout << "PAPI ERROR" << endl;
		return;
	}
	cout << real_time << "\t" << (flops/real_time)/1e9;


	return 0;





}



