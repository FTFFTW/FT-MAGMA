#include "hip/hip_runtime.h"
/*
    Enhanced Online ABFT
    UC Riverside
    Jieyang Chen
*/
#include "FT.h"
#include "common_magma.h"
#include "magma.h"
#include <stdlib.h>

#define NB 4
// encoding checksum for A
#define B 32
#define rB 32
#define cB 32
#define N 30720

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	/* logrithm reduction */
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


__global__ void
chkenc_kernel2(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * NB + threadIdx.x;

	A = A + idx * lda;

	double temp = 0;
	double temp2 = 0;
	for (int i = 0; i < NB; i++) {
		temp += A[i];
		temp2 += A[i] * (i+1);
	}
	*(Chk + idx * ldchk) = temp;
	*(Chk + idx * ldchk+1) = temp2;
	
}


__global__ void
chkenc_kernel3(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * B;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	__shared__ double cache[B][B];

	for (int i = 0; i < NB; i += B) {
		
		//load a block to cache
		for (int j = 0; j < B; j++) {
			cache[threadIdx.x][j] = *(A + j * lda + threadIdx.x);
			//if (blockIdx.x == 0 && threadIdx.x == 0) {
			//	printf("%f ", cache[threadIdx.x][j]);
			//}
		}

		//if (blockIdx.x == 0 && threadIdx.x == 0) {
		//		printf("\n");
	    //}
		__syncthreads();

		for (int j = 0; j < B; j++) {
			sum1 += cache[j][threadIdx.x];
			sum2 += cache[j][threadIdx.x] * (i + j + 1);
			//if (blockIdx.x == 0 && threadIdx.x == 0) {
			//	printf("%f ", cache[j][threadIdx.x]);
			//}
		}
		//if (blockIdx.x == 0 && threadIdx.x == 0) {
		//		printf("\n");
	    //}
		__syncthreads();
		A = A + B;
	}

	idx += threadIdx.x;

	*(Chk + idx * ldchk) = sum1;
	*(Chk + idx * ldchk+1) = sum2;
	
}




void chkenc(double * A, int lda, int m, int n, double * Chk , int ldchk, magma_queue_t stream) {
  /*  int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 32; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel4, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	//printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );
	*/
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	chkenc_kernel<<<n, NB, 0, stream>>>(A, lda, Chk, ldchk);

}



