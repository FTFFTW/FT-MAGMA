#include "hip/hip_runtime.h"
/*
    Enhanced Online ABFT
    UC Riverside
    Jieyang Chen
*/
#include "FT.h"
#include "common_magma.h"
#include "magma.h"
#include <stdlib.h>

#define NB 512
// encoding checksum for A

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{
	//if (blockIdx.x == 0 && threadIdx.x == 0)
	//	printf("grid:%d, block:%d\n", gridDim.x, blockDim.x);
    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	int i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	int i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


void chkenc(double * A, int lda, int m, int n, double * Chk , int ldchk, magma_queue_t stream) {
	chkenc_kernel<<<n, m>>>(A, lda, Chk, ldchk);
}



