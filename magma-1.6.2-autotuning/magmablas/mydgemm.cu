#include "hip/hip_runtime.h"
/*
    Enhanced Online ABFT
    UC Riverside
    Jieyang Chen
*/
#include "FT.h"
#include "common_magma.h"
#include "magma.h"
#include <stdlib.h>

#define NB 512
// encoding checksum for A
#define B 32
#define rB 8
#define cB 64
#define N 30720

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	/* logrithm reduction */
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


__global__ void
chkenc_kernel1_5(double * A, int lda, double * Chk , int ldchk)
{

	//blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();


	double sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk) = sum;
	}

	__syncthreads();

	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();


	sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk + 1) = sum;
	}
	
}

__global__ void
chkenc_kernel2(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * NB + threadIdx.x;

	A = A + idx * lda;

	double temp = 0;
	double temp2 = 0;
	for (int i = 0; i < NB; i++) {
		temp += A[i];
		temp2 += A[i] * (i+1);
	}
	*(Chk + idx * ldchk) = temp;
	*(Chk + idx * ldchk+1) = temp2;
	
}


__global__ void
chkenc_kernel3(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * cB;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	__shared__ double cache[rB][cB];

	for (int i = 0; i < NB; i += rB) {
		
		//load a block to cache
		if (threadIdx.x < rB) {
			for (int j = 0; j < cB; j++) {
				cache[threadIdx.x][j] = *(A + j * lda + threadIdx.x);
				//cache[j][threadIdx.x] = *(A + j * lda + threadIdx.x);
			}
		}

		__syncthreads();

		for (int j = 0; j < rB; j++) {
			sum1 += cache[j][threadIdx.x];
			sum2 += cache[j][threadIdx.x] * (i + j + 1);
			
		}
		
		__syncthreads();

		A = A + B;
	}

	idx += threadIdx.x;

	*(Chk + idx * ldchk) = sum1;
	*(Chk + idx * ldchk+1) = sum2;
	
}

__global__ void
chkenc_kernel3_5(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    

    int rb = blockDim.x;
    int cb = blockDim.y; 

    int idx = blockIdx.x * cb;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	extern __shared__ double cache[]; //rB * cB

	for (int i = 0; i < NB; i += rb) {
		
		//load a block to cache
		cache[threadIdx.x + threadIdx.y * rb] = *(A + threadIdx.y * lda + threadIdx.x);
		__syncthreads();
		int k = rb / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.x + threadIdx.y * rb] += cache[threadIdx.x + k + threadIdx.y * rb];
			}
			
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum1 += cache[0 + threadIdx.y * rb];
		}

		cache[threadIdx.x + threadIdx.y * rb] = *(A + threadIdx.y * lda + threadIdx.x) * (i + threadIdx.x + 1);
		__syncthreads();
		k = rb / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.x + threadIdx.y * rb] += cache[threadIdx.x + k + threadIdx.y * rb];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum2 += cache[0 + threadIdx.y * rb];
		}
				
		A = A + rb;
	}

	idx += threadIdx.y;

	if (threadIdx.x == 0) {
		*(Chk + idx * ldchk) = sum1;
		*(Chk + idx * ldchk+1) = sum2;
	}
	
}



void chkenc(double * A, int lda, int m, int n, double * chk , int ldchk, magma_queue_t stream) {
  /*  int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 32; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel4, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	//printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );
	*/
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	//int rb = 8;
	//int cb = 16;
	//dim3 d(rb, cb, 1);
	//chkenc_kernel3_5<<<N/cb, d, rb*cb*sizeof(double), stream>>>(A, lda, chk, ldchk);
	chkenc_kernel3<<<n/cB, cB, 0, stream>>>(A, lda, chk, ldchk);

}



